#include "hip/hip_runtime.h"
#include "module_hsolver/include/dngvd_op.h"


#include <hipsolver.h>

namespace hsolver
{

template <>
void dngvx_op<double, psi::DEVICE_GPU>::operator()(const psi::DEVICE_GPU* d,
                                                   const int row,
                                                   const int col,
                                                   const std::complex<double>* A,
                                                   const std::complex<double>* B,
                                                   const int m,
                                                   double* W,
                                                   std::complex<double>* V)
{
    createBLAShandle();
    // init A_eigenvectors, transpose_B and all_W
    double2 *A_eigenvectors, *transpose_B;
    hipMalloc((void**)&A_eigenvectors, sizeof(double2) * row * col);
    hipMalloc((void**)&transpose_B, sizeof(double2) * row * col);

    matrixTranspose_op<double, psi::DEVICE_GPU>()(d, col, row, A, (std::complex<double>*)A_eigenvectors);
    matrixTranspose_op<double, psi::DEVICE_GPU>()(d, col, row, B, (std::complex<double>*)transpose_B);




    double* all_W;
    hipMalloc((void**)&all_W, sizeof(double) * col);

    // prepare some values for hipsolverDnZhegvd_bufferSize
    hipsolverHandle_t cusolverH;
    hipsolverDnCreate(&cusolverH);
    int* devInfo;
    hipMalloc((void**)&devInfo, sizeof(int));

    // calculate the sizes needed for pre-allocated buffer.
    int lwork = 0;
    hipsolverDnZhegvd_bufferSize(
        cusolverH,
        HIPSOLVER_EIG_TYPE_1, // itype = HIPSOLVER_EIG_TYPE_1: A*x = (lambda)*B*x.
        HIPSOLVER_EIG_MODE_VECTOR, // jobz = HIPSOLVER_EIG_MODE_VECTOR : Compute eigenvalues and eigenvectors.
        HIPBLAS_FILL_MODE_LOWER,
        col,
        A_eigenvectors,
        // (double2)A,
        row,
        transpose_B,
        // (double2)B,
        row,
        all_W,
        &lwork);

    // allocate memery
    hipDoubleComplex* d_work;
    hipMalloc((void**)&d_work, sizeof(hipDoubleComplex) * lwork);

    // compute eigenvalues and eigenvectors.
    hipsolverDnZhegvd(
        cusolverH,
        HIPSOLVER_EIG_TYPE_1, // itype = HIPSOLVER_EIG_TYPE_1: A*x = (lambda)*B*x.
        HIPSOLVER_EIG_MODE_VECTOR, // jobz = HIPSOLVER_EIG_MODE_VECTOR : Compute eigenvalues and eigenvectors.
        HIPBLAS_FILL_MODE_LOWER,
        col,
        A_eigenvectors,
        // (double2)A,
        row,
        transpose_B,
        // (double2)B,
        row,
        all_W,
        d_work,
        lwork,
        devInfo);

    hipDeviceSynchronize();

    // get eigenvalues and eigenvectors.  only m !
    hipMemcpy(W, all_W, sizeof(double) * m, hipMemcpyDeviceToDevice);

    hipMemcpy(V, A_eigenvectors, sizeof(std::complex<double>)*col*m, hipMemcpyDeviceToDevice);

    matrixTranspose_op<double, psi::DEVICE_GPU>()(d, col, row, V, V);

    int info_gpu;
    hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    assert(0 == info_gpu);
    
    // free the buffer
    hipFree(d_work);
    // free resources and destroy
    hipFree(A_eigenvectors);
    hipFree(all_W);
    hipFree(devInfo);
    hipsolverDnDestroy(cusolverH);
    destoryBLAShandle();
}

template <>
void dngv_op<double, psi::DEVICE_GPU>::operator()(const psi::DEVICE_GPU* d,
                                                  const int row,
                                                  const int col,
                                                  const std::complex<double>* A,
                                                  const std::complex<double>* B,
                                                  double* W,
                                                  std::complex<double>* V)
{
    createBLAShandle();
    // init A_eigenvectors & transpose_B
    double2 *A_eigenvectors, *transpose_B;
    hipMalloc((void**)&A_eigenvectors, sizeof(double2) * row * col);
    hipMalloc((void**)&transpose_B, sizeof(double2) * row * col);
    
    // transpose A to A_eigenvectors
    matrixTranspose_op<double, psi::DEVICE_GPU>()(d, row, col, A, (std::complex<double>*)A_eigenvectors);
    // transpose B to transpose_B
    matrixTranspose_op<double, psi::DEVICE_GPU>()(d, row, col, B, (std::complex<double>*)transpose_B);
    
    // init all_W
    double* all_W;
    hipMalloc((void**)&all_W, sizeof(double) * row);

    // prepare some values for hipsolverDnZhegvd_bufferSize
    hipsolverHandle_t cusolverH;
    hipsolverDnCreate(&cusolverH);
    int* devInfo;
    hipMalloc((void**)&devInfo, sizeof(int));

    // calculate the sizes needed for pre-allocated buffer.
    int lwork = 0;
    hipsolverDnZhegvd_bufferSize(
        cusolverH,
        HIPSOLVER_EIG_TYPE_1, // itype = HIPSOLVER_EIG_TYPE_1: A*x = (lambda)*B*x.
        HIPSOLVER_EIG_MODE_VECTOR, // jobz = HIPSOLVER_EIG_MODE_VECTOR : Compute eigenvalues and eigenvectors.
        HIPBLAS_FILL_MODE_UPPER,
        row,
        A_eigenvectors,
        col,
        transpose_B,
        col,
        all_W,
        &lwork);

    // allocate memery
    hipDoubleComplex* d_work;
    hipMalloc((void**)&d_work, sizeof(hipDoubleComplex) * lwork);

    // compute eigenvalues and eigenvectors.
    hipsolverDnZhegvd(
        cusolverH,
        HIPSOLVER_EIG_TYPE_1, // itype = HIPSOLVER_EIG_TYPE_1: A*x = (lambda)*B*x.
        HIPSOLVER_EIG_MODE_VECTOR, // jobz = HIPSOLVER_EIG_MODE_VECTOR : Compute eigenvalues and eigenvectors.
        HIPBLAS_FILL_MODE_UPPER,
        row,
        A_eigenvectors,
        col,
        transpose_B,
        col,
        all_W,
        d_work,
        lwork,
        devInfo);

    hipDeviceSynchronize();

    // get all eigenvalues and eigenvectors.
    hipMemcpy(W, all_W, sizeof(double) * row, hipMemcpyDeviceToDevice);
    matrixTranspose_op<double, psi::DEVICE_GPU>()(d, row, col, (std::complex<double>*)A_eigenvectors, V);

    int info_gpu;
    hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    assert(0 == info_gpu);

    // free the buffer
    hipFree(d_work);
    // free resources and destroy
    hipFree(A_eigenvectors);
    hipFree(all_W);
    hipFree(devInfo);
    hipsolverDnDestroy(cusolverH);
    destoryBLAShandle();
}

} // namespace hsolver