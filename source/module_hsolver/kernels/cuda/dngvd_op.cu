#include "hip/hip_runtime.h"
#include "module_hsolver/kernels/dngvd_op.h"
#include "hip/hip_runtime_api.h"

#include <hipsolver.h>

#define cusolverErrcheck(res)                      \
    {                                              \
        cusolverAssert((res), __FILE__, __LINE__); \
    }

// cuSOLVER API errors
static const char* _cusolverGetErrorEnum(hipsolverStatus_t error)
{
    switch (error)
    {
    case HIPSOLVER_STATUS_SUCCESS:
        return "HIPSOLVER_STATUS_SUCCESS";
    case HIPSOLVER_STATUS_NOT_INITIALIZED:
        return "HIPSOLVER_STATUS_NOT_INITIALIZED";
    case HIPSOLVER_STATUS_ALLOC_FAILED:
        return "HIPSOLVER_STATUS_ALLOC_FAILED";
    case HIPSOLVER_STATUS_INVALID_VALUE:
        return "HIPSOLVER_STATUS_INVALID_VALUE";
    case HIPSOLVER_STATUS_ARCH_MISMATCH:
        return "HIPSOLVER_STATUS_ARCH_MISMATCH";
    case HIPSOLVER_STATUS_MAPPING_ERROR:
        return "HIPSOLVER_STATUS_MAPPING_ERROR";
    case HIPSOLVER_STATUS_EXECUTION_FAILED:
        return "HIPSOLVER_STATUS_EXECUTION_FAILED";
    case HIPSOLVER_STATUS_INTERNAL_ERROR:
        return "HIPSOLVER_STATUS_INTERNAL_ERROR";
    case HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
        return "HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
    case HIPSOLVER_STATUS_NOT_SUPPORTED:
        return "HIPSOLVER_STATUS_NOT_SUPPORTED ";
    case HIPSOLVER_STATUS_ZERO_PIVOT:
        return "HIPSOLVER_STATUS_ZERO_PIVOT";
    case CUSOLVER_STATUS_INVALID_LICENSE:
        return "CUSOLVER_STATUS_INVALID_LICENSE";
    }
    return "<unknown>";
}

template<typename FPTYPE>
hipDataType get_cuda_data_type (FPTYPE * /* datatype */);

template<>
hipDataType get_cuda_data_type (float * /* datatype */) {
    return HIP_C_32F;
}

template<>
hipDataType get_cuda_data_type (double * /* datatype */) {
    return HIP_C_64F;
}


inline void cusolverAssert(hipsolverStatus_t code, const char* file, int line, bool abort = true)
{
    if (code != HIPSOLVER_STATUS_SUCCESS)
    {
        fprintf(stderr, "cuSOLVER Assert: %s %s %d\n", _cusolverGetErrorEnum(code), file, line);
        if (abort)
            exit(code);
    }
}

namespace hsolver
{

static hipsolverHandle_t cusolver_H = nullptr;

void createCUSOLVERhandle()
{
    if (cusolver_H == nullptr)
    {
        cusolverErrcheck(hipsolverDnCreate(&cusolver_H));
    }
}

void destoryCUSOLVERhandle()
{
    if (cusolver_H != nullptr)
    {
        cusolverErrcheck(hipsolverDnDestroy(cusolver_H));
        cusolver_H = nullptr;
    }
}

static inline
void xpotrf_wapper (const hipblasFillMode_t& uplo, const int& n, std::complex<float> * A, const int& lda)
{
    int lwork;
    hipsolverDnCpotrf_bufferSize(cusolver_H, uplo, n, reinterpret_cast<float2 *>(A), n, &lwork);
    float2* work;
    hipMalloc((void**)&work, lwork * sizeof(float2));
    // Perform Cholesky decomposition
    hipsolverDnCpotrf(cusolver_H, uplo, n, reinterpret_cast<float2 *>(A), n, work, lwork, nullptr);
    hipFree(work);
}

static inline
void xpotrf_wapper (const hipblasFillMode_t& uplo, const int& n, std::complex<double> * A, const int& lda)
{
    int lwork;
    hipsolverDnZpotrf_bufferSize(cusolver_H, uplo, n, reinterpret_cast<double2 *>(A), n, &lwork);
    double2* work;
    hipMalloc((void**)&work, lwork * sizeof(double2));
    // Perform Cholesky decomposition
    hipsolverDnZpotrf(cusolver_H, uplo, n, reinterpret_cast<double2 *>(A), n, work, lwork, nullptr);
    hipFree(work);
}

static inline
void xhegvd_wrapper (
        const hipblasFillMode_t& uplo,
        const int& n,
        std::complex<float> * A, const int& lda,
        std::complex<float> * B, const int& ldb,
        float * W)
{
    // prepare some values for hipsolverDnZhegvd_bufferSize
    int * devInfo = nullptr;
    int lwork = 0, info_gpu = 0;
    float2 * work = nullptr;
    checkCudaErrors(hipMalloc((void**)&devInfo, sizeof(int)));

    // calculate the sizes needed for pre-allocated buffer.
    cusolverErrcheck(hipsolverDnChegvd_bufferSize(cusolver_H, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, uplo, n,
                                                 reinterpret_cast<const float2 *>(A), lda,
                                                 reinterpret_cast<const float2 *>(B), ldb, W, &lwork));
    // allocate memery
    checkCudaErrors(hipMalloc((void**)&work, sizeof(float2) * lwork));

    // compute eigenvalues and eigenvectors.
    cusolverErrcheck(hipsolverDnChegvd(cusolver_H, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, uplo, n,
                                      reinterpret_cast<float2 *>(A), lda, reinterpret_cast<float2 *>(B), ldb, W, work, lwork, devInfo));

    checkCudaErrors(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    assert(0 == info_gpu);
    // free the buffer
    checkCudaErrors(hipFree(work));
    checkCudaErrors(hipFree(devInfo));
}

static inline
void xhegvd_wrapper (
        const hipblasFillMode_t& uplo,
        const int& n,
        std::complex<double> * A, const int& lda,
        std::complex<double> * B, const int& ldb,
        double * W)
{
    // prepare some values for hipsolverDnZhegvd_bufferSize
    int * devInfo = nullptr;
    int lwork = 0, info_gpu = 0;
    double2 * work = nullptr;
    checkCudaErrors(hipMalloc((void**)&devInfo, sizeof(int)));

    // calculate the sizes needed for pre-allocated buffer.
    cusolverErrcheck(hipsolverDnZhegvd_bufferSize(cusolver_H, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, uplo, n,
                                                 reinterpret_cast<const double2 *>(A), lda,
                                                 reinterpret_cast<const double2 *>(B), ldb, W, &lwork));
    // allocate memery
    checkCudaErrors(hipMalloc((void**)&work, sizeof(double2) * lwork));

    // compute eigenvalues and eigenvectors.
    cusolverErrcheck(hipsolverDnZhegvd(cusolver_H, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, uplo, n,
                                      reinterpret_cast<double2 *>(A), lda, reinterpret_cast<double2 *>(B), ldb, W, work, lwork, devInfo));

    checkCudaErrors(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    assert(0 == info_gpu);
    // free the buffer
    checkCudaErrors(hipFree(work));
    checkCudaErrors(hipFree(devInfo));
}

static inline
void xheevd_wrapper (
        const hipblasFillMode_t& uplo,
        const int& n,
        std::complex<float> * A, const int& lda,
        float * W)
{
    // prepare some values for hipsolverDnZhegvd_bufferSize
    int * devInfo = nullptr;
    int lwork = 0, info_gpu = 0;
    float2 * work = nullptr;
    checkCudaErrors(hipMalloc((void**)&devInfo, sizeof(int)));

    // calculate the sizes needed for pre-allocated buffer.
    cusolverErrcheck(hipsolverDnCheevd_bufferSize(cusolver_H, HIPSOLVER_EIG_MODE_VECTOR, uplo, n,
                                                 reinterpret_cast<const float2 *>(A), lda, W, &lwork));
    // allocate memery
    checkCudaErrors(hipMalloc((void**)&work, sizeof(float2) * lwork));
    // compute eigenvalues and eigenvectors.
    cusolverErrcheck(hipsolverDnCheevd(cusolver_H, HIPSOLVER_EIG_MODE_VECTOR, uplo, n, reinterpret_cast<float2 *>(A), lda, W, work, lwork, devInfo));

    checkCudaErrors(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    assert(0 == info_gpu);
    checkCudaErrors(hipFree(work));
    checkCudaErrors(hipFree(devInfo));
}

static inline
void xheevd_wrapper (
        const hipblasFillMode_t& uplo,
        const int& n,
        std::complex<double> * A, const int& lda,
        double * W)
{
    // prepare some values for hipsolverDnZhegvd_bufferSize
    int * devInfo = nullptr;
    int lwork = 0, info_gpu = 0;
    double2 * work = nullptr;
    checkCudaErrors(hipMalloc((void**)&devInfo, sizeof(int)));

    // calculate the sizes needed for pre-allocated buffer.
    cusolverErrcheck(hipsolverDnZheevd_bufferSize(cusolver_H, HIPSOLVER_EIG_MODE_VECTOR, uplo, n,
                                                 reinterpret_cast<const double2 *>(A), lda, W, &lwork));
    // allocate memery
    checkCudaErrors(hipMalloc((void**)&work, sizeof(double2) * lwork));
    // compute eigenvalues and eigenvectors.
    cusolverErrcheck(hipsolverDnZheevd(cusolver_H, HIPSOLVER_EIG_MODE_VECTOR, uplo, n,
                                      reinterpret_cast<double2 *>(A), lda, W, work, lwork, devInfo));

    checkCudaErrors(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    assert(0 == info_gpu);
    checkCudaErrors(hipFree(work));
    checkCudaErrors(hipFree(devInfo));
}

template <typename FPTYPE>
struct dngvd_op<FPTYPE, psi::DEVICE_GPU> {
    void operator()(
            const psi::DEVICE_GPU *d,
            const int nstart,
            const int ldh,
            const std::complex<FPTYPE> *A, // hcc
            const std::complex<FPTYPE> *B, // scc
            FPTYPE *W, // eigenvalue
            std::complex<FPTYPE> *V)
    {
        assert(nstart == ldh);
        // A to V
        checkCudaErrors(hipMemcpy(V, A, sizeof(std::complex<FPTYPE>) * ldh * nstart, hipMemcpyDeviceToDevice));
        xhegvd_wrapper(HIPBLAS_FILL_MODE_UPPER, nstart, V, ldh,
                       (std::complex<FPTYPE> *)B, ldh, W);
    }
};

template <typename FPTYPE>
struct dnevx_op<FPTYPE, psi::DEVICE_GPU> {
    void operator()(
            const psi::DEVICE_GPU *d,
            const int nstart,
            const int ldh,
            const std::complex<FPTYPE> *A, // hcc
            const int m,
            FPTYPE *W, // eigenvalue
            std::complex<FPTYPE> *V)
    {
        assert(nstart <= ldh);
        // A to V
        checkCudaErrors(hipMemcpy(V, A, sizeof(std::complex<FPTYPE>) * nstart * ldh, hipMemcpyDeviceToDevice));
        xheevd_wrapper(HIPBLAS_FILL_MODE_LOWER, nstart, V, ldh, W);
    }
};

template <typename FPTYPE>
struct dnevd_op<FPTYPE, psi::DEVICE_GPU> {
    void operator()(const psi::DEVICE_GPU* dev, std::complex<FPTYPE>* A, const int& dim, FPTYPE* W)
    {
        // A to V
        xheevd_wrapper(HIPBLAS_FILL_MODE_UPPER, dim, A, dim, W);
    }
};

template <typename FPTYPE>
struct zpotrf_op<FPTYPE, psi::DEVICE_GPU> {
    void operator()(const psi::DEVICE_GPU* /*dev*/,std::complex<FPTYPE>* A, const int& dim)
    {
        xpotrf_wapper(HIPBLAS_FILL_MODE_UPPER, dim, A, dim);
    }
};

template <typename FPTYPE>
struct ztrtri_op<FPTYPE, psi::DEVICE_GPU> {
    void operator()(const psi::DEVICE_GPU* /*dev*/,std::complex<FPTYPE>* A, const int& dim) {
        FPTYPE *d_work = nullptr;
        size_t d_lwork = 0;
        FPTYPE *h_work = nullptr;
        size_t h_lwork = 0;

        cusolverDnXtrtri_bufferSize(cusolver_H, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_DIAG_NON_UNIT, dim,
                                    get_cuda_data_type<FPTYPE>(d_work),
                                    reinterpret_cast<void *>(A), dim, &d_lwork,
                                    &h_lwork);

        hipMalloc(reinterpret_cast<void **>(&d_work), d_lwork);

        if (h_lwork) {
            h_work = (FPTYPE *)malloc(h_lwork);
        }

        cusolverDnXtrtri(cusolver_H, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_DIAG_NON_UNIT, dim, get_cuda_data_type<FPTYPE>(d_work),
                         reinterpret_cast<void *>(A), dim,
                         d_work, d_lwork, h_work, h_lwork, nullptr);

        hipFree(d_work);
        free(h_work);
    }
};

template struct zpotrf_op<float, psi::DEVICE_GPU>;
template struct ztrtri_op<float, psi::DEVICE_GPU>;
template struct dngvd_op<float, psi::DEVICE_GPU>;
template struct dnevd_op<float, psi::DEVICE_GPU>;
template struct dnevx_op<float, psi::DEVICE_GPU>;
template struct dngvd_op<double, psi::DEVICE_GPU>;
template struct dnevx_op<double, psi::DEVICE_GPU>;
template struct dnevd_op<double, psi::DEVICE_GPU>;
template struct zpotrf_op<double, psi::DEVICE_GPU>;
template struct ztrtri_op<double, psi::DEVICE_GPU>;

} // namespace hsolver